#include "hip/hip_runtime.h"
#include "carve.h"

#define CUDA_BLOCKS 56
#define THREADS_PER_BLOCK 256

__global__ void signature_match_h(unsigned char *buffer, filerec_d *d_results, uint64_t offset,
                                  uint64_t blocksize, unsigned char *d_header,
                                  size_t *d_header_offsets, size_t *d_header_size,
                                  int *d_header_type, ssize_t ARR_SIZE, uint64_t num_blocks,
                                  uint64_t processed, int *global_counter) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_blocks) {
                return;
        }

        uint64_t already_processed = (processed * blocksize);
        uint64_t block_start = idx * blocksize;

        for (size_t i = 0; i < blocksize; i++) {
                uint64_t current_pos = block_start + i;

                for (size_t j = 0; j < ARR_SIZE; j++) {
                        uint64_t d_strt = d_header_offsets[j];
                        size_t pattern_size = d_header_size[j];
                        bool match = true;

                        if (current_pos + pattern_size > (idx + 1) * blocksize) {
                                continue;
                        }
                        for (size_t k = 0; k < pattern_size; k++) {
                                if (buffer[current_pos + k] != d_header[d_strt + k]) {
                                        match = false;
                                        break;
                                }
                        }

                        if (match) {
                                int result_idx = atomicAdd(global_counter, 1);

                                d_results[result_idx].type = d_header_type[j];
                                d_results[result_idx].location =
                                    current_pos + offset + already_processed;
                                d_results[result_idx].sign_type = 1;
                        }
                }
        }
}

__global__ void signature_match_f(unsigned char *buffer, filerec_d *d_results, uint64_t offset,
                                  uint64_t blocksize, unsigned char *d_footer,
                                  size_t *d_footer_offsets, size_t *d_footer_size,
                                  int *d_footer_type, ssize_t ARR_SIZE, uint64_t num_blocks,
                                  uint64_t processed, int *global_counter) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= num_blocks) {
                return;
        }

        uint64_t already_processed = (processed * blocksize);
        uint64_t block_start = idx * blocksize;

        for (size_t i = 0; i < blocksize; i++) {
                uint64_t current_pos = block_start + i;

                for (size_t j = 0; j < ARR_SIZE; j++) {
                        if(d_footer_size[j] == 0) {
                                continue;
                        } 
                        uint64_t d_strt = d_footer_offsets[j];
                        size_t pattern_size = d_footer_size[j];
                        bool match = true;

                        if (current_pos + pattern_size > (idx + 1) * blocksize) {
                                continue;
                        }
                        for (size_t k = 0; k < pattern_size; k++) {
                                if (buffer[current_pos + k] != d_footer[d_strt + k]) {
                                        match = false;
                                        break;
                                }
                        }

                        if (match) {
                                int result_idx = atomicAdd(global_counter, 1);

                                d_results[result_idx].type = d_footer_type[j];
                                d_results[result_idx].location =
                                    current_pos + offset + already_processed;
                                d_results[result_idx].sign_type = 2;
                        }
                }
        }
}

void CudaErr(hipError_t error) {
        if (error != hipSuccess) {
                std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
                exit(EXIT_FAILURE);
        }
}

int d_carve(xfs_fskey &key, std::vector<filerec_d> &fr, xfs_superblock &sb, std::ifstream &fs,
            std::vector<filetype> &ft) {
        // Check Cuda
        hipError_t cudaStatus;
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
                std::cout << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"
                          << std::endl;
        }

        hipError_t err;

        // Initialize Useful Variables
        uint64_t agblocks = ntohl(sb.sb_agblocks);
        uint64_t blocksize = ntohl(sb.sb_blocksize);

        uint64_t agno = key.k_agno;
        uint64_t start = key.k_start;
        uint64_t count = key.k_count;

        uint64_t blk_offset = agblk_offset(agno, sb);
        uint64_t offset = blk_offset + (start * blocksize);
        uint64_t total_blocks = count;
        const uint64_t batch_size = CUDA_BLOCKS * THREADS_PER_BLOCK;

        fs.seekg(0, fs.end);
        size_t eof = fs.tellg();

        //  Counter
        int *d_header_counter;
        hipMalloc(&d_header_counter, sizeof(int));
        hipMemset(d_header_counter, 0, sizeof(int));

        int *d_footer_counter;
        hipMalloc(&d_footer_counter, sizeof(int));
        hipMemset(d_footer_counter, 0, sizeof(int));

        // Creating Header Arrays
        size_t total_header_size = 0;
        size_t h_offset = 0;
        std::vector<size_t> host_header_sizes(ft.size());
        std::vector<int> host_header_types(ft.size());
        for (size_t i = 0; i < ft.size(); ++i) {
                if (ft[i].Header_s != 0) {
                        total_header_size += ft[i].Header_s;
                        host_header_sizes[i] = ft[i].Header_s;
                        host_header_types[i] = ft[i].index;
                }
        }

        unsigned char *h_flat_headers = new unsigned char[total_header_size];
        size_t header_offsets[ft.size() * sizeof(size_t)];

        for (size_t i = 0; i < ft.size(); ++i) {
                if (ft[i].Header_s != 0) {
                        header_offsets[i] = h_offset;
                        std::memcpy(h_flat_headers + h_offset, ft[i].Header.data(), ft[i].Header_s);
                        h_offset += ft[i].Header_s;
                }
        }

        // Creating Footer Arrays
        size_t total_footer_size = 0;
        size_t f_offset = 0;
        std::vector<size_t> host_footer_sizes(ft.size());
        std::vector<int> host_footer_types(ft.size());
        for (size_t i = 0; i < ft.size(); ++i) {
                if (ft[i].Footer_s != 0) {
                        total_footer_size += ft[i].Footer_s;
                        host_footer_sizes[i] = ft[i].Footer_s;
                        host_footer_types[i] = ft[i].index;
                }
        }

        unsigned char *h_flat_footer = new unsigned char[total_footer_size];
        size_t footer_offsets[ft.size() * sizeof(size_t)];

        for (size_t i = 0; i < ft.size(); ++i) {
                if (ft[i].Footer_s != 0) {
                        footer_offsets[i] = f_offset;
                        std::memcpy(h_flat_footer + f_offset, ft[i].Footer.data(), ft[i].Footer_s);
                        f_offset += ft[i].Footer_s;
                }
        }

        // Device buffers
        unsigned char *d_buffer;
        filerec_d *h_results;
        filerec_d *f_results;

        unsigned char *d_flat_headers;
        size_t *d_header_offsets;
        size_t *d_header_size;
        int *d_header_type;

        unsigned char *d_flat_footers;
        size_t *d_footer_offsets;
        size_t *d_footer_size;
        int *d_footer_type;

        // Allocating Device Buffers
        err = hipMalloc(&d_buffer, (batch_size * blocksize));
        CudaErr(err);

        size_t max_results = total_blocks * ft.size();
        err = hipMalloc(&h_results, max_results * sizeof(filerec_d));
        CudaErr(err);
        err = hipMalloc(&f_results, max_results * sizeof(filerec_d));
        CudaErr(err);
        err = hipMalloc(&d_flat_headers, total_header_size);
        CudaErr(err);
        err = hipMalloc(&d_header_offsets, ft.size() * sizeof(size_t));
        CudaErr(err);
        err = hipMalloc(&d_header_size, ft.size() * sizeof(size_t));
        CudaErr(err);
        err = hipMalloc(&d_header_type, ft.size() * sizeof(int));
        CudaErr(err);

        err = hipMalloc(&d_flat_footers, total_footer_size);
        CudaErr(err);
        err = hipMalloc(&d_footer_offsets, ft.size() * sizeof(size_t));
        CudaErr(err);
        err = hipMalloc(&d_footer_size, ft.size() * sizeof(size_t));
        CudaErr(err);
        err = hipMalloc(&d_footer_type, ft.size() * sizeof(int));
        CudaErr(err);

        // Host -> Device Copying
        err = hipMemcpy(d_flat_headers, h_flat_headers, total_header_size, hipMemcpyHostToDevice);
        CudaErr(err);
        err = hipMemcpy(d_header_size, host_header_sizes.data(), ft.size() * sizeof(size_t),
                         hipMemcpyHostToDevice);
        CudaErr(err);
        err = hipMemcpy(d_header_type, host_header_types.data(), ft.size() * sizeof(int),
                         hipMemcpyHostToDevice);
        CudaErr(err);
        err = hipMemcpy(d_header_offsets, header_offsets, ft.size() * sizeof(size_t),
                         hipMemcpyHostToDevice);
        CudaErr(err);

        err = hipMemcpy(d_flat_footers, h_flat_footer, total_footer_size, hipMemcpyHostToDevice);
        CudaErr(err);
        err = hipMemcpy(d_footer_size, host_footer_sizes.data(), ft.size() * sizeof(size_t),
                         hipMemcpyHostToDevice);
        CudaErr(err);
        err = hipMemcpy(d_footer_type, host_footer_types.data(), ft.size() * sizeof(int),
                         hipMemcpyHostToDevice);
        CudaErr(err);
        err = hipMemcpy(d_footer_offsets, footer_offsets, ft.size() * sizeof(size_t),
                         hipMemcpyHostToDevice);
        CudaErr(err);

        err = hipMemset(h_results, 0, total_blocks * sizeof(filerec_d));
        CudaErr(err);
        err = hipMemset(f_results, 0, total_blocks * sizeof(filerec_d));
        CudaErr(err);

        // Main Host Buffer
        char *host_buffer = new char[blocksize * batch_size];

        // Loop analyze batch_size over free space
        for (uint64_t processed_blocks = 0; processed_blocks < total_blocks;
             processed_blocks += batch_size) {
                uint64_t current_batch_size = batch_size;
                if (processed_blocks + batch_size > total_blocks) {
                        current_batch_size = total_blocks - processed_blocks;
                }
                uint64_t r_size;

                // Filesystem Read to host buffer
                int64_t seek_pos = offset + (processed_blocks * blocksize);
                fs.seekg(seek_pos, fs.beg);
                if (offset + (processed_blocks * blocksize) + (blocksize * current_batch_size) >
                    eof) {
                        r_size = eof - (offset + (processed_blocks * blocksize));
                } else {
                        r_size = (blocksize * current_batch_size);
                }

                fs.read(host_buffer, r_size);
                if (!fs.good()) {
                        std::cout << "Error: Read Error" << std::endl;
                        return -1;
                }

                // Host -> Device main buffer
                err = hipMemcpy(d_buffer, host_buffer, r_size, hipMemcpyHostToDevice);
                CudaErr(err);

                // Kernel
                signature_match_h<<<CUDA_BLOCKS, THREADS_PER_BLOCK>>>(
                    d_buffer, h_results, offset, blocksize, d_flat_headers, d_header_offsets,
                    d_header_size, d_header_type, ft.size(), current_batch_size, processed_blocks,
                    d_header_counter);
                hipDeviceSynchronize();
                // signature_match_f<<<CUDA_BLOCKS, THREADS_PER_BLOCK>>>(
                //     d_buffer, f_results, offset, blocksize, d_flat_footers, d_footer_offsets,
                //     d_footer_size, d_footer_type, ft.size(), current_batch_size, processed_blocks,
                //     d_footer_counter);
                // hipDeviceSynchronize();

                // Host Vector to store results
                int header_count, footer_count;
                hipMemcpy(&header_count, d_header_counter, sizeof(int), hipMemcpyDeviceToHost);
                // hipMemcpy(&footer_count, d_footer_counter, sizeof(int), hipMemcpyDeviceToHost);

                std::vector<filerec_d> header_results(header_count);
                // std::vector<filerec_d> /* footer_results */(footer_count);

                err = hipMemcpy(header_results.data(), h_results, header_count * sizeof(filerec_d),
                                 hipMemcpyDeviceToHost);
                CudaErr(err);
                // err = hipMemcpy(footer_results.data(), f_results, footer_count * sizeof(filerec_d),
                                 // hipMemcpyDeviceToHost);
                // CudaErr(err);
                for (int i = 0; i < header_count; i++) {
                        fr.push_back(header_results[i]);
                }

                // for (int i = 0; i < footer_count; i++) {
                        // fr.push_back(footer_results[i]);
                // }
        }
        hipFree(d_buffer);
        hipFree(h_results);
        hipFree(f_results);
        hipFree(d_flat_headers);
        hipFree(d_header_offsets);
        hipFree(d_header_size);
        hipFree(d_header_type);
        hipFree(d_flat_footers);
        hipFree(d_footer_offsets);
        hipFree(d_footer_size);
        hipFree(d_footer_type);
        hipFree(d_header_counter);
        hipFree(d_footer_counter);
        delete[] h_flat_headers;
        delete[] h_flat_footer;
        delete[] host_buffer;

        return 0;
}
